#include "hip/hip_runtime.h"
#include "hashtable.cuh"
#include "errorcheck.h"
#include "lock.cuh"
#include <stdio.h>

__global__
void init_table::init_empty_table(Data * table, int size) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    while (i < size) {
		auto ptr = table + i;
		ptr->lock.init();
		ptr->key = 0;
		ptr->state = EMPTY;
		i += gridDim.x * blockDim.x;
	}
}

__device__
int HashFunction::h1(LL key, int size) {
	return key % size;
}

__device__
int HashFunction::h2(LL key, int size) {
	return 1;
}

HashTable::HashTable(int size) {
	this->size = size;
	gpuErrchk( hipMalloc(&table, size * sizeof(Data)) );
	int threads_per_block = 32,
		blocks = (size/threads_per_block) + (size % threads_per_block != 0);
	init_table::init_empty_table<<<blocks, threads_per_block>>>(table, size);
}

__device__
bool HashTable::insert(LL key) {
	int N = this->size, h1 = HashFunction::h1(key, size), h2 = HashFunction::h2(key, size);
	int index = h1;
	while(N-->0){
		auto current = (table+index);
		State s = current->state;
		if(s != FULL){
			if( current->lock.lock() ) {
				if(s != FULL) {
					current->state = FULL;
					current->key = key;
					current->lock.unlock();
					return true;
					// Can't guarantee that the element will be there after insert returns...
				}
			}
		}
		index += h2;
	}
	return false;
}

void HashTable::check() {
	Data * hostTable = new Data[size];
	gpuErrchk( hipMemcpy(hostTable, table, size * sizeof(Data), hipMemcpyDeviceToHost) );
	for(int i = 0; i < size; ++i) {
		if(!(hostTable+i)->lock.trylock()) {
			printf("Hashtable locks not initialized properly!!\n");
			break;
		}
		else if((hostTable+i)->key != 0 || (hostTable+i)->state != EMPTY){
			printf("That's weird...");
			break;
		}
	}
	delete [] hostTable;
	printf("yay!!\n");
}

HashTable::~HashTable() {
	gpuErrchk( hipFree(table) );
}