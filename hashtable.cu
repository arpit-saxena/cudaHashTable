#include "hip/hip_runtime.h"
#include "hashtable.cuh"
#include "errorcheck.h"
#include "lock.cuh"
#include <stdio.h>

__global__
void init_table::init_empty_table(Data * table, int size) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    while (i < size) {
		auto ptr = table + i;
		ptr->lock.init();
		ptr->key = 0;
		ptr->state = EMPTY;
		i += gridDim.x * blockDim.x;
	}
}

HashTable::HashTable(int size) {
	this->size = size;
	gpuErrchk( hipMalloc(&table, size * sizeof(Data)) );
	int threads_per_block = 32,
		blocks = (size/threads_per_block) + (size % threads_per_block != 0);
	init_table::init_empty_table<<<blocks, threads_per_block>>>(table, size);
}

__device__
bool HashTable::insert(LL key){
	return true;
}

void HashTable::check() {
	Data * hostTable = new Data[size];
	gpuErrchk( hipMemcpy(hostTable, table, size * sizeof(Data), hipMemcpyDeviceToHost) );
	for(int i = 0; i < size; ++i) {
		if(!(hostTable+i)->lock.trylock()) {
			printf("Hashtable locks not initialized properly!!\n");
			break;
		}
		else if((hostTable+i)->key != 0 || (hostTable+i)->state != EMPTY){
			printf("That's weird...");
			break;
		}
	}
	delete [] hostTable;
	printf("yay!!\n");
}

HashTable::~HashTable() {
	gpuErrchk( hipFree(table) );
}