#include "hip/hip_runtime.h"
#include "lock.cuh"
#include "errorcheck.h"
#include "hashtable.cuh"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <utility>

__global__
void testKernel(Lock* locks, int num_locks) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    Lock *lock = locks + id % num_locks;

    while (true) {
        if (lock -> lock(Thread::Insert)) {
            printf("[%d] Locked %d\n", id, id % num_locks);

            if (lock -> unlock()) {
                printf("[%d] Unlocked %d\n", id, id % num_locks);
                break;
            } else {
                printf("[%d] ERROR: Not able to unlock %d\n", id, id % num_locks);
            }
        } else {
            printf("[%d] Failed to lock %d\n", id, id % num_locks);
        }
    }
}

__global__
void initLocks(Lock *locks, int num_locks) {
    for (
        int i = blockIdx.x * blockDim.x + threadIdx.x; 
        i < num_locks;
        i += gridDim.x * blockDim.x
    ) {
        locks[i].init();
    }
}

void checkLocks() {
    Lock *locks;
    int num_locks = 2;

    gpuErrchk( hipMalloc(&locks, num_locks * sizeof(Lock)) );
    initLocks<<<1, num_locks>>>(locks, num_locks);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    testKernel<<<1, 4>>>(locks, num_locks);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk( hipFree(locks) );

    HashTable h1(64);
    h1.check();
}

std::pair<Instruction *, int> getInstructions(std::string name) {
    std::ifstream fin(name);
    int numIns; fin >> numIns;
    Instruction *ins = (Instruction *) malloc(numIns * sizeof(Instruction));
    for (int i = 0; i < numIns; i++) {
        std::string type; fin >> type;
        if (type == "INSERT") {
            ins[i].type = Instruction::Insert;
        } else if (type == "DELETE") {
            ins[i].type = Instruction::Delete;
        } else if (type == "FIND") {
            ins[i].type = Instruction::Find;
        } else {
            printf("Undefined instruction %s\n", type.c_str());
        }

        LL key; fin >> key;
        ins[i].key = key;
    }
    return std::make_pair(ins, numIns);
}

int main() {
    HashTable table(100);
    auto p = getInstructions("instructions.txt");
    Instruction *ins = p.first;
    int numIns = p.second;

    Instruction *d_ins;
    gpuErrchk( hipMalloc(&d_ins, numIns * sizeof(Instruction)) );
    gpuErrchk( hipMemcpy(d_ins, ins, numIns * sizeof(Instruction), hipMemcpyDefault) );

    HashTable::performInstructs(table, d_ins, numIns, nullptr);
    HashTable::print(table);

    gpuErrchk( hipDeviceSynchronize() );
    free(ins);
    hipFree(d_ins);

    return 0;
}