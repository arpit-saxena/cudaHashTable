#include "hip/hip_runtime.h"
#include "lock.cuh"
#include "errorcheck.h"
#include "hashtable.cuh"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <utility>
#include <vector>
#include <chrono>
#include <ctime>

const int hashTable_size = 20;
const bool logging = false;

__global__
void testKernel(Lock* locks, int num_locks) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    Lock *lock = locks + id % num_locks;

    while (true) {
        if (lock -> lock(Thread::Insert)) {
            printf("[%d] Locked %d\n", id, id % num_locks);

            if (lock -> unlock()) {
                printf("[%d] Unlocked %d\n", id, id % num_locks);
                break;
            } else {
                printf("[%d] ERROR: Not able to unlock %d\n", id, id % num_locks);
            }
        } else {
            printf("[%d] Failed to lock %d\n", id, id % num_locks);
        }
    }
}

__global__
void initLocks(Lock *locks, int num_locks) {
    for (
        int i = blockIdx.x * blockDim.x + threadIdx.x; 
        i < num_locks;
        i += gridDim.x * blockDim.x
    ) {
        locks[i].init();
    }
}

void checkLocks() {
    Lock *locks;
    int num_locks = 2;

    gpuErrchk( hipMalloc(&locks, num_locks * sizeof(Lock)) );
    initLocks<<<1, num_locks>>>(locks, num_locks);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    testKernel<<<1, 4>>>(locks, num_locks);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk( hipFree(locks) );
}

using std::vector;

vector<vector<Instruction>> getInstructions(std::string name) {
    std::ifstream fin(name);
    int numBlocks; fin >> numBlocks;
    vector<vector<Instruction>> instructions(numBlocks);
    for(int blockNum = 0; blockNum < numBlocks; blockNum++) {
        int numIns; fin >> numIns;
        for (int i = 0; i < numIns; i++) {
            Instruction ins;
            std::string type; fin >> type;
            if (type == "INSERT") {
                ins.type = Instruction::Insert;
            } else if (type == "DELETE") {
                ins.type = Instruction::Delete;
            } else if (type == "FIND") {
                ins.type = Instruction::Find;
            } else {
                printf("Undefined instruction %s\n", type.c_str());
            }

            ULL key; fin >> key;
            ins.key = key;
            instructions[blockNum].push_back(ins);
        }
    }
    return instructions;
}

int main(int argc, char **argv) {
    HashTable h_table(hashTable_size);
    HashTable *table;
    gpuErrchk( hipMalloc(&table, sizeof(HashTable)) );
    gpuErrchk( hipMemcpy(table, &h_table, sizeof(HashTable), hipMemcpyHostToDevice) );

    auto p = getInstructions(argv[1]);

    std::ofstream fout;
    if (logging) fout.open("log.txt");
    for (auto &v_ins : p) {
        int numIns = v_ins.size();
        Instruction *ins = (Instruction *) malloc(sizeof(Instruction) * numIns);
        std::copy(v_ins.begin(), v_ins.end(), ins);

        Instruction *d_ins;
        gpuErrchk( hipMalloc(&d_ins, numIns * sizeof(Instruction)) );
        gpuErrchk( hipMemcpy(d_ins, ins, numIns * sizeof(Instruction), hipMemcpyDefault) );

        ThreadLog * statuses = nullptr;
        if (logging) {
            statuses = (ThreadLog *)malloc(sizeof(ThreadLog)*numIns);
            for(int i = 0; i < numIns; ++i) {
                new (statuses + i) ThreadLog(h_table.size, ins[i]);
            }
        }

        hipEvent_t start;
        gpuErrchk( hipEventCreate(&start) );

        hipEvent_t stop;
        gpuErrchk( hipEventCreate(&stop) );

        gpuErrchk( hipEventRecord(start, NULL) );
        HashTable::performInstructs(table, d_ins, numIns, statuses);
        gpuErrchk( hipEventRecord(stop, NULL) );

        gpuErrchk( hipEventSynchronize(stop) );
        float msecTotal = 0.0f;
        gpuErrchk( hipEventElapsedTime(&msecTotal, start, stop) );

        auto time_now = std::chrono::system_clock::now();
        auto time = std::chrono::system_clock::to_time_t(time_now);
        if (statuses)
            HashTable::print(table, statuses, numIns, fout << "\n\nLogged at: " << std::ctime(&time));
        std::cout << "Time taken by performInstructs: " << msecTotal << " ms" << std::endl;

        gpuErrchk( hipDeviceSynchronize() );
        free(ins);
        hipFree(d_ins);
        for(int i = 0; i < numIns; ++i) {
            (statuses + i)->~ThreadLog();
        }
    }

    hipFree(table);
    return 0;
}